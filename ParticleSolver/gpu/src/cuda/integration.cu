#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hiprand.h>
#include <stdio.h>

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>

#include "hip/hip_runtime_api.h"
#include "integration_kernel.cuh"
#include "util.cuh"
#include "shared_variables.cuh"

//#define PRINT

hiprandGenerator_t gen(0);

thrust::device_vector<float> V; // particle velocities
thrust::device_vector<float> lambda;
thrust::device_vector<float> denom;

thrust::device_vector<float> ros;

thrust::device_vector<uint> neighbors;
thrust::device_vector<uint> numNeighbors;

thrust::device_vector<float> textureVec;

float *rands;

extern "C"
{
    /*****************************************************************************
     *                              INITIALIZATION
     *****************************************************************************/


    void initIntegration()
    {
        allocateArray((void **)&rands, 6 * sizeof(float));
        checkCudaErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
        checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));
    }

    void appendIntegrationParticle(float *v, float *ro, uint numParticles)
    {
        int sizeV = V.size();
        int sizeRo = ros.size();

        // resize the vectors
        V.resize(sizeV + 4 * numParticles);
        ros.resize(sizeRo + numParticles);

        // get raw pointers to the data
        float *dV = thrust::raw_pointer_cast(V.data());
        float *dRos = thrust::raw_pointer_cast(ros.data());

        // copy the new data over to the gpu
        copyArrayToDevice(dV + sizeV, v, 0, 4 * numParticles * sizeof(float));
        copyArrayToDevice(dRos + sizeRo, ro, 0, numParticles * sizeof(float));

        // resize but don't need to fill
        lambda.resize(ros.size());
        numNeighbors.resize(ros.size());
        neighbors.resize(V.size() * MAX_FLUID_NEIGHBORS);
        textureVec.resize(V.size());
    }

    void freeIntegrationVectors()
    {
         V.clear();
         lambda.clear();
         denom.clear();
         ros.clear();
         neighbors.clear();
         numNeighbors.clear();
         textureVec.clear();

         V.shrink_to_fit();
         lambda.shrink_to_fit();
         denom.shrink_to_fit();
         ros.shrink_to_fit();
         neighbors.shrink_to_fit();
         numNeighbors.shrink_to_fit();
         textureVec.shrink_to_fit();

         checkCudaErrors(hiprandDestroyGenerator(gen));
         freeArray(rands);
    }

    void setParameters(SimParams *hostParams)
    {
        // copy parameters to constant memory
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, 1 * sizeof(SimParams)));
    }

















    /*****************************************************************************
     *                              UPDATE POSITIONS
     *****************************************************************************/

    void integrateSystem(float *pos, float deltaTime, uint numParticles)
    {
        thrust::device_ptr<float4> d_pos4((float4 *)pos);
        thrust::device_ptr<float4> d_vel4((float4 *)thrust::raw_pointer_cast(V.data()));

        // copy current positions for reference later
        copyToXstar(pos, numParticles);

        // guess new positions based on forces
        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_vel4+numParticles)),
            integrate_functor(deltaTime));
    }





















    /*****************************************************************************
     *                              BUILD GRID
     *****************************************************************************/

    void calcHash(uint *gridParticleHash, uint *gridParticleIndex, float *pos, int numParticles)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // execute the kernel
        calcHashD<<< numBlocks, numThreads >>>(gridParticleHash, gridParticleIndex, (float4 *) pos, numParticles);

#ifdef PRINT
        printf("HASHES:\n");
        thrust::device_ptr<uint> dGPH(gridParticleHash);
        for (uint i = 0; i < numParticles; i++)
        {
            printf("particle: %u: hash: %u\n", i, (uint)*(dGPH + i));
        }
        printf("\n");
#endif

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }


    void reorderDataAndFindCellStart(uint  *cellStart,
                                     uint  *cellEnd,
                                     float *sortedPos,
                                     float *sortedW,
                                     int   *sortedPhase,
                                     uint  *gridParticleHash,
                                     uint  *gridParticleIndex,
                                     float *oldPos,
                                     uint   numParticles,
                                     uint   numCells)
    {
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        // set all cells to empty
        checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

        float *dW = getWRawPtr();
        int *dPhase = getPhaseRawPtr();

        thrust::device_ptr<float> d_pos(oldPos);
        thrust::device_ptr<float> d_W(dW);
        thrust::device_ptr<int> d_Phase(dPhase);

//        printf("particles\n");
//        for (int i = 0; i < numParticles; i++)
//        {
//            printf("%.2f %.2f %.2f %.2f, ", (float)*(d_pos+i*4), (float)*(d_pos+i*4+1), (float)*(d_pos+i*4+2), (float)*(d_pos+i*4+3));
//            printf("w: %.2f, phase: %d\n", (float)*(d_W+i), (int)*(d_Phase+i));
//        }
//        printf("\n");
//        float4 *pos4 = (float4*)oldPos;
        float *pos;
        checkCudaErrors(hipMalloc((void**)&pos, numParticles*4*sizeof(float)));
//        textureVec.resize(4 * numParticles);
//        float *pos = thrust::raw_pointer_cast(textureVec.data());
        checkCudaErrors(hipMemcpy(pos, oldPos, numParticles*4*sizeof(float), hipMemcpyDeviceToDevice));

        checkCudaErrors(hipBindTexture(0, oldPosTex, pos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, invMassTex, dW, numParticles*sizeof(float)));
        checkCudaErrors(hipBindTexture(0, oldPhaseTex, dPhase, numParticles*sizeof(int)));

        uint smemSize = sizeof(uint)*(numThreads+1);
        reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(cellStart,
                                                                           cellEnd,
                                                                           (float4 *) sortedPos,
                                                                           sortedW,
                                                                           sortedPhase,
                                                                           gridParticleHash,
                                                                           gridParticleIndex,
                                                                           (float4 *) oldPos,
                                                                           dW,
                                                                           dPhase,
                                                                           numParticles);
        getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");

#ifdef PRINT
        printf("Sorted:\n");
        thrust::device_ptr<uint> dGPH(gridParticleHash);
        thrust::device_ptr<uint> dGPI(gridParticleIndex);
        for (uint i = 0; i < numParticles; i++)
        {
            printf("i: %u: hash: %u\n", i, (uint)*(dGPH + i));
            printf("i: %u: part: %u\n", i, (uint)*(dGPI + i));
        }
        printf("\n");


        printf("Sorted:\n");
        thrust::device_ptr<uint> dstart(cellStart);
        thrust::device_ptr<uint> dend(cellEnd);
        for (uint i = 0; i < 16; i++)
        {
            printf("i: %u: start: %u\n", i, (uint)*(dstart + i));
            printf("i: %u: end: %u\n", i, (uint)*(dend + i));
        }
        printf("\n");
#endif

        checkCudaErrors(hipUnbindTexture(oldPosTex));
        checkCudaErrors(hipUnbindTexture(invMassTex));
        checkCudaErrors(hipUnbindTexture(oldPhaseTex));

        checkCudaErrors(hipFree(pos));
    }

    void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
    {
        thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
                            thrust::device_ptr<uint>(dGridParticleHash + numParticles),
                            thrust::device_ptr<uint>(dGridParticleIndex));
    }


































    /*****************************************************************************
     *                              PROCESS COLLISIONS
     *****************************************************************************/

    void sortByType(float *dPos, uint numParticles)
    {

    }

    void collideWorld(float *pos, float *sortedPos, uint numParticles, int3 minBounds, int3 maxBounds)
    {
        thrust::device_ptr<float4> d_pos4((float4 *)pos);
        thrust::device_ptr<float4> d_Xstar((float4*)getXstarRawPtr());
        thrust::device_ptr<int> d_phase(getPhaseRawPtr());

        // create random vars for boundary collisions
        checkCudaErrors(hiprandGenerateUniform(gen, rands, 6));

        // check for boundary collisions and move particles
//        thrust::for_each
//        thrust::transform(d_pos4, d_pos4 + numParticles, d_Xstar, d_pos4, collide_world_functor(rands, minBounds, maxBounds));

        thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_Xstar, d_phase)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_Xstar+numParticles, d_phase+numParticles)),
            collide_world_functor(rands, minBounds, maxBounds));
    }

    void collide(float *particles,
                 float *sortedPos,
                 float *sortedW,
                 int   *sortedPhase,
                 uint  *gridParticleIndex,
                 uint  *cellStart,
                 uint  *cellEnd,
                 uint   numParticles,
                 uint   numCells)
    {
        checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, invMassTex, sortedW, numParticles*sizeof(float)));
        checkCudaErrors(hipBindTexture(0, oldPhaseTex, sortedPhase, numParticles*sizeof(int)));

        checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
        checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));

        // store neighbors
        uint *dNeighbors = thrust::raw_pointer_cast(neighbors.data());
        uint *dNumNeighbors = thrust::raw_pointer_cast(numNeighbors.data());
        float *dXstar = getXstarRawPtr();

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 64, numBlocks, numThreads);

        // execute the kernel
        collideD<<< numBlocks, numThreads >>>((float4 *)particles,
                                              (float4 *)dXstar,
                                              (float4 *)sortedPos,
                                              sortedW,
                                              sortedPhase,
                                              gridParticleIndex,
                                              cellStart,
                                              cellEnd,
                                              numParticles,
                                              dNeighbors,
                                              dNumNeighbors);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

        checkCudaErrors(hipUnbindTexture(oldPosTex));
        checkCudaErrors(hipUnbindTexture(invMassTex));
        checkCudaErrors(hipUnbindTexture(oldPhaseTex));

        checkCudaErrors(hipUnbindTexture(cellStartTex));
        checkCudaErrors(hipUnbindTexture(cellEndTex));
    }

























    /*****************************************************************************
     *                              UPDATE VELOCITIES
     *****************************************************************************/

    void calcVelocity(float *dpos, float deltaTime, uint numParticles)
    {
        float *dXstar = getXstarRawPtr();
        thrust::device_ptr<float4> d_Xstar((float4*)dXstar);
        thrust::device_ptr<float4> d_pos((float4*)dpos);
        thrust::device_ptr<float4> d_vel((float4*)thrust::raw_pointer_cast(V.data()));


        thrust::transform(d_pos, d_pos + numParticles, d_Xstar, d_vel, subtract_functor(deltaTime));

    }























    /*****************************************************************************
     *                              SOLVE FLUIDS
     *****************************************************************************/
    void solveFluids(float *sortedPos,
                     float *sortedW,
                     int   *sortedPhase,
                     uint  *gridParticleIndex,
                     uint  *cellStart,
                     uint  *cellEnd,
                     float *particles,
                     uint   numParticles,
                     uint   numCells)
    {
        checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, invMassTex, sortedW, numParticles*sizeof(float)));
        checkCudaErrors(hipBindTexture(0, oldPhaseTex, sortedPhase, numParticles*sizeof(float4)));
        checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
        checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));

        // thread per particle
        uint numThreads, numBlocks;
        computeGridSize(numParticles, 256, numBlocks, numThreads);

        float *dLambda = thrust::raw_pointer_cast(lambda.data());
//        float *dDenom = thrust::raw_pointer_cast(denom.data());
        uint *dNeighbors = thrust::raw_pointer_cast(neighbors.data());
        uint *dNumNeighbors = thrust::raw_pointer_cast(numNeighbors.data());
        float *dRos = thrust::raw_pointer_cast(ros.data());

//        printf("ros: %u, numParts: %u\n", (uint)ros.size(), numParticles);

        // execute the kernel
        findLambdasD<<< numBlocks, numThreads >>>(dLambda,
                                                  gridParticleIndex,
                                                  cellStart,
                                                  cellEnd,
                                                  numParticles,
                                                  dNeighbors,
                                                  dNumNeighbors,
                                                  dRos);

        // execute the kernel
        solveFluidsD<<< numBlocks, numThreads >>>(dLambda,
                                                  gridParticleIndex,
                                                  (float4 *) particles,
                                                  numParticles,
                                                  dNeighbors,
                                                  dNumNeighbors,
                                                  dRos);

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");

        checkCudaErrors(hipUnbindTexture(oldPosTex));
        checkCudaErrors(hipUnbindTexture(invMassTex));
        checkCudaErrors(hipUnbindTexture(oldPhaseTex));
        checkCudaErrors(hipUnbindTexture(cellStartTex));
        checkCudaErrors(hipUnbindTexture(cellEndTex));
    }
}
